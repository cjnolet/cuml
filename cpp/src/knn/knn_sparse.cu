/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cumlHandle.hpp>

#include <cuml/common/logger.hpp>
#include <cuml/neighbors/knn_sparse.hpp>

#include <sparse/knn.cuh>

#include <hipsparse.h>

namespace ML {
namespace Sparse {

void brute_force_knn(cumlHandle &handle, const int *idxIndptr,
                     const int *idxIndices, const float *idxData, size_t idxNNZ,
                     size_t n_idx_rows, size_t n_idx_cols,
                     const int *queryIndptr, const int *queryIndices,
                     const float *queryData, size_t queryNNZ,
                     size_t n_query_rows, size_t n_query_cols,
                     int *output_indices, float *output_dists, int k,
                     size_t batch_size,  // approx 1M
                     ML::MetricType metric, float metricArg,
                     bool expanded_form) {
  std::shared_ptr<deviceAllocator> d_alloc = handle.getDeviceAllocator();
  hipsparseHandle_t cusparse_handle = handle.getImpl().getcusparseHandle();
  hipStream_t stream = handle.getStream();

  MLCommon::Sparse::Selection::brute_force_knn(
    idxIndptr, idxIndices, idxData, idxNNZ, n_idx_rows, n_idx_cols, queryIndptr,
    queryIndices, queryData, queryNNZ, n_query_rows, n_query_cols,
    output_indices, output_dists, k, cusparse_handle, d_alloc, stream,
    batch_size, metric, metricArg, expanded_form);
}
};  // namespace Sparse
};  // namespace ML
