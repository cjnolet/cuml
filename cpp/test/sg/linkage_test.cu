/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/cuda_utils.cuh>
#include <vector>

#include <cuml/cluster/linkage.hpp>
#include <cuml/common/cuml_allocator.hpp>
#include <cuml/cuml.hpp>
#include <cuml/datasets/make_blobs.hpp>
#include <cuml/metrics/metrics.hpp>


#include <raft/linalg/distance_type.h>
#include <raft/linalg/cublas_wrappers.h>
#include <raft/linalg/transpose.h>

#include <test_utils.h>

#include <common/device_buffer.hpp>
#include <cuml/common/logger.hpp>

namespace ML {

using namespace MLCommon;
using namespace Datasets;
using namespace Metrics;
using namespace std;

template <typename T, typename IdxT>
struct LinkageInputs {
  IdxT n_row;
  IdxT n_col;
  IdxT n_centers;
  T cluster_std;
  unsigned long long int seed;
};

template <typename T, typename IdxT>
::std::ostream &operator<<(::std::ostream &os,
                           const LinkageInputs<T, IdxT> &dims) {
  return os;
}

template <typename T, typename IdxT>
class LinkageTest : public ::testing::TestWithParam<LinkageInputs<T, IdxT>> {
 protected:
  void basicTest() {
    raft::handle_t handle;

    params = ::testing::TestWithParam<LinkageInputs<T, IdxT>>::GetParam();

    device_buffer<T> out(handle.get_device_allocator(), handle.get_stream(),
                         params.n_row * params.n_col);
    device_buffer<IdxT> l(handle.get_device_allocator(), handle.get_stream(),
                          params.n_row);

    make_blobs(handle, out.data(), l.data(), params.n_row, params.n_col,
               params.n_centers, true, nullptr, nullptr, params.cluster_std,
               true, -10.0f, 10.0f, params.seed);

    raft::allocate(labels, params.n_row);
    raft::allocate(labels_ref, params.n_row);

    raft::copy(labels_ref, l.data(), params.n_row, handle.get_stream());

    CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));


    ML::linkage_output<IdxT, T> out_arrs;

    device_buffer<IdxT> out_children(handle.get_device_allocator(), handle.get_stream(),
                                  (params.n_row - 1) * 2);
    out_arrs.children = out_children.data();
    out_arrs.labels = l.data();

    ML::single_linkage(handle,
                       out.data(),
                       params.n_row,
                       params.n_col,
                       raft::distance::DistanceType::EucUnexpandedL2,
                       &out_arrs);


    CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));

    score = adjusted_rand_index(handle, labels_ref, labels, params.n_row);

    if (score < 1.0) {
      auto str = raft::arr2Str(labels_ref, params.n_row, "labels_ref",
                               handle.get_stream());
      CUML_LOG_DEBUG("y: %s", str.c_str());
      str = raft::arr2Str(labels, params.n_row, "labels", handle.get_stream());
      CUML_LOG_DEBUG("y_hat: %s", str.c_str());
      CUML_LOG_DEBUG("Score = %lf", score);
    }
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(labels_ref));
  }

 protected:
  LinkageInputs<T, IdxT> params;
  IdxT *labels, *labels_ref;

  double score;
};

const std::vector<LinkageInputs<float, int>> inputsf2 = {
  {500, 16, 5, 0.01,  1234ULL},
  {1000, 1000, 10, 0.01, 1234ULL},
  {20000, 10000, 10, 0.01, 1234ULL},
  {20000, 100, 5000, 0.01, 1234ULL}};

typedef LinkageTest<float, int> LinkageTestF_Int;
TEST_P(LinkageTestF_Int, Result) { ASSERT_TRUE(score == 1.0); }

INSTANTIATE_TEST_CASE_P(LinkageTests, LinkageTestF_Int,
                        ::testing::ValuesIn(inputsf2));

template <typename T>
struct Linkage2DArrayInputs {
  const T *points;
  const int *out;
  size_t n_row;
  // n_out allows to compare less labels than we have inputs
  // (some output labels can be ambiguous)
  size_t n_out;
};

template <typename T>
class Linkage2DSimple : public ::testing::TestWithParam<Linkage2DArrayInputs<T>> {
 protected:
  void basicTest() {
    raft::handle_t handle;

    params = ::testing::TestWithParam<Linkage2DArrayInputs<T>>::GetParam();

    raft::allocate(inputs, params.n_row * 2);
    raft::allocate(labels, params.n_row);
    raft::allocate(labels_ref, params.n_out);
    raft::allocate(core_sample_indices_d, params.n_row);

    raft::copy(inputs, params.points, params.n_row * 2, handle.get_stream());
    raft::copy(labels_ref, params.out, params.n_out, handle.get_stream());
    CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));

    ML::linkage_output<int, T> out_arrs;

    device_buffer<int> out_children(handle.get_device_allocator(), handle.get_stream(),
                                     (params.n_row - 1) * 2);
    out_arrs.children = out_children.data();
    out_arrs.labels = labels;

    ML::single_linkage(handle,
                       inputs,
                       params.n_row,
                       2,
                       raft::distance::DistanceType::EucUnexpandedL2,
                       &out_arrs);



    CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));

    score = adjusted_rand_index(handle, labels_ref, labels, (int)params.n_out);

    if (score < 1.0) {
      auto str = raft::arr2Str(labels_ref, params.n_out, "labels_ref",
                               handle.get_stream());
      CUML_LOG_DEBUG("y: %s", str.c_str());
      str = raft::arr2Str(labels, params.n_row, "labels", handle.get_stream());
      CUML_LOG_DEBUG("y_hat: %s", str.c_str());
      CUML_LOG_DEBUG("Score = %lf", score);
    }
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    CUDA_CHECK(hipFree(labels_ref));
    CUDA_CHECK(hipFree(labels));
    CUDA_CHECK(hipFree(inputs));
    CUDA_CHECK(hipFree(core_sample_indices_d));
  }

 protected:
  Linkage2DArrayInputs<T> params;
  int *labels, *labels_ref;
  int *core_sample_indices_d;
  T *inputs;

  double score;
};

// The input looks like a latin cross or a star with a chain:
//   .
// . . . . .
//   .
// There is 1 core-point (intersection of the bars)
// and the two points to the very right are not reachable from it
// So there should be one cluster (the plus/star on the left)
// and two noise points
const std::vector<float> test2d1_f = {0,  0, 1, 0, 1, 1, 1,
                                      -1, 2, 0, 3, 0, 4, 0};
const std::vector<double> test2d1_d(test2d1_f.begin(), test2d1_f.end());
const std::vector<int> test2d1_l = {0, 0, 0, 0, 0, -1, -1};
const std::vector<int> test2d1c_l = {1, -1, -1, -1, -1, -1, -1};

// The input looks like a long two-barred (orhodox) cross or
// two stars next to each other:
//   .     .
// . . . . . .
//   .     .
// There are 2 core-points but they are not reachable from each other
// So there should be two clusters, both in the form of a plus/star
const std::vector<float> test2d2_f = {0, 0, 1, 0, 1, 1, 1, -1, 2, 0,
                                      3, 0, 4, 0, 4, 1, 4, -1, 5, 0};
const std::vector<double> test2d2_d(test2d2_f.begin(), test2d2_f.end());
const std::vector<int> test2d2_l = {0, 0, 0, 0, 0, 1, 1, 1, 1, 1};
const std::vector<int> test2d2c_l = {1, 6, -1, -1, -1, -1, -1, -1, -1, -1};

// The input looks like a two-barred (orhodox) cross or
// two stars sharing a link:
//   .   .
// . . . . .
//   .   .
// There are 2 core-points but they are not reachable from each other
// So there should be two clusters.
// However, the link that is shared between the stars
// actually has an ambiguous label (to the best of my knowledge)
// as it will depend on the order in which we process the core-points.
// Note that there are 9 input points, but only 8 labels for this reason
const std::vector<float> test2d3_f = {
  0, 0, 1, 0, 1, 1, 1, -1, 3, 0, 3, 1, 3, -1, 4, 0, 2, 0,
};
const std::vector<double> test2d3_d(test2d3_f.begin(), test2d3_f.end());
const std::vector<int> test2d3_l = {0, 0, 0, 0, 1, 1, 1, 1};
const std::vector<int> test2d3c_l = {1, 4, -1, -1, -1, -1, -1, -1, -1};

const std::vector<Linkage2DArrayInputs<float>> inputs2d_f = {
  {test2d1_f.data(), test2d1_l.data(), test2d1_f.size() / 2, test2d1_l.size()},
  {test2d2_f.data(), test2d2_l.data(), test2d2_f.size() / 2, test2d2_l.size()},
  {test2d3_f.data(), test2d3_l.data(), test2d3_f.size() / 2, test2d3_l.size()},
};


typedef Linkage2DSimple<float> Linkage2DSimple_F;
TEST_P(Linkage2DSimple_F, Result) { ASSERT_TRUE(score == 1.0); }

INSTANTIATE_TEST_CASE_P(LinkageTests, Linkage2DSimple_F,
                        ::testing::ValuesIn(inputs2d_f));

}  // end namespace ML
