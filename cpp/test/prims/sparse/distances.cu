/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <gtest/gtest.h>
#include <sparse/distances.cuh>
#include "test_utils.h"
#include <hipsparse.h>
#include <raft/sparse/cusparse_wrappers.h>

namespace MLCommon {
namespace Sparse {
namespace Selection {


template <typename value_idx, typename value_t>
struct DistancesInputs {};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const DistancesInputs<value_idx, value_t> &dims) {
  return os;
}


template <typename value_idx, typename value_t>
class DistancesTest : public ::testing::TestWithParam<DistancesInputs<value_idx, value_t>> {
 protected:

	void make_data() {
		std::vector<value_idx> indptr_h = { 0, 2, 4, 6, 8 };
		std::vector<value_idx> indices_h = { 0, 1, 0, 1, 0, 1, 0, 1 };
		std::vector<value_t> data_h = { 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f };

		allocate(indptr, 5);
		allocate(indices, 8);
		allocate(data, 8);

		updateDevice(indptr, indptr_h.data(), 5, stream);
		updateDevice(indices, indices_h.data(), 8, stream);
		updateDevice(data, data_h.data(), 8, stream);

		allocate(csc_indptr, 3);
		allocate(csc_indices, 8);

		std::vector<value_idx> csc_indptr_h = { 0, 4, 8 };
		std::vector<value_idx> csc_indices_h = { 0, 1, 2, 3, 0, 1, 2, 3 };

		updateDevice(csc_indptr, csc_indptr_h.data(), 3, stream);
		updateDevice(csc_indices, csc_indices_h.data(), 8, stream);

		out_indptr_h = { 0, 2, 4, 6, 8 };
		out_indices_h = { 0, 1, 0, 1, 0, 1, 0, 1 };
		out_data_h = { 2.0f, 2.0f, 2.0f, 2.0f, 2.0f, 2.0f, 2.0f, 2.0f };
	}



  void SetUp() override {
    params = ::testing::TestWithParam<DistancesInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(new defaultDeviceAllocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));


    make_data();

    std::cout << "Done making data. Running dists" << std::endl;

    Distance::distances_config_t<value_idx, value_t>
      dist_config;
    dist_config.index_nrows = 4;
    dist_config.index_ncols = 2;
    dist_config.index_nnz = 8;
    dist_config.csc_index_indptr = csc_indptr;
    dist_config.csc_index_indices = csc_indices;
    dist_config.csc_index_data = data;
    dist_config.search_nrows = 4;
    dist_config.search_ncols = 2;
    dist_config.search_nnz = 8;
    dist_config.csr_search_indptr = indptr;
    dist_config.csr_search_indices = indices;
    dist_config.csr_search_data = data;
    dist_config.handle = cusparseHandle;
    dist_config.allocator = alloc;
    dist_config.stream = stream;

    allocate(out_indptr, 5);

    Distance::ip_distances_t<value_idx, value_t> compute_dists(dist_config);
    value_idx out_nnz = compute_dists.get_nnz(out_indptr);

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "New NNZ: " << out_nnz << std::endl;

    allocate(out_indices, out_nnz);
    allocate(out_data, out_nnz);

    std::cout << "Running compute()" << std::endl;

    compute_dists.compute(out_indptr, out_indices, out_data);

    std::cout << "Synchronizing" << std::endl;

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "Done running dists" << std::endl;

  }

  void TearDown() override {

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(csc_indptr));
    CUDA_CHECK(hipFree(csc_indices));
    CUDA_CHECK(hipFree(out_indptr));
    CUDA_CHECK(hipFree(out_indices));
    CUDA_CHECK(hipFree(out_data));
  }

  void compare() {

	  std::cout << arr2Str(out_indptr, 5, "out_indptr", stream) << std::endl;
	  std::cout << arr2Str(out_indices, 16, "out_indices", stream) << std::endl;
	  std::cout << arr2Str(out_data, 16, "out_data", stream) << std::endl;
  }

 protected:

  hipStream_t stream;
  hipsparseHandle_t cusparseHandle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // transposed input
  value_idx *csc_indptr, *csc_indices;

  // output data
  value_idx *out_indptr, *out_indices;
  value_t *out_data;

  std::vector<value_idx> out_indptr_h;
  std::vector<value_idx> out_indices_h;
  std::vector<value_t> out_data_h;

  DistancesInputs<value_idx, value_t> params;
};

const std::vector<DistancesInputs<int, float>> inputs_i32_f = {{}};
typedef DistancesTest<int, float> DistancesTestF;
TEST_P(DistancesTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(DistancesTests, DistancesTestF, ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Selection
};  // end namespace Sparse
};  // end namespace MLCommon
