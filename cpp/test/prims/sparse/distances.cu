/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <hipsparse.h>
#include <gtest/gtest.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <sparse/distances.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Sparse {
namespace Selection {

template <typename value_idx, typename value_t>
struct DistancesInputs {};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const DistancesInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class DistancesTest
  : public ::testing::TestWithParam<DistancesInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = {0, 2, 4, 6, 8};
    std::vector<value_idx> indices_h = {0, 1, 0, 1, 0, 1, 0, 1};
    std::vector<value_t> data_h = {1.0f, 2.0f, 1.0f, 2.0f,
                                   1.0f, 2.0f, 1.0f, 2.0f};

    allocate(indptr, 5);
    allocate(indices, 8);
    allocate(data, 8);

    updateDevice(indptr, indptr_h.data(), 5, stream);
    updateDevice(indices, indices_h.data(), 8, stream);
    updateDevice(data, data_h.data(), 8, stream);

    allocate(csc_indptr, 3);
    allocate(csc_indices, 8);
    allocate(csc_data, 8);

    std::vector<value_idx> csc_indptr_h = {0, 4, 8};
    std::vector<value_idx> csc_indices_h = {0, 1, 2, 3, 0, 1, 2, 3};
    std::vector<value_t> csc_data_h = {1.0f, 1.0f, 1.0f, 1.0f,
                                       2.0f, 2.0f, 2.0f, 2.0f};

    updateDevice(csc_indptr, csc_indptr_h.data(), 3, stream);
    updateDevice(csc_indices, csc_indices_h.data(), 8, stream);
    updateDevice(csc_data, csc_data_h.data(), 8, stream);

    std::vector<value_t> out_dists_ref_h = {
      5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0,
      5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0,
    };

    allocate(out_dists_ref, 16);

    updateDevice(out_dists_ref, out_dists_ref_h.data(), out_dists_ref_h.size(),
                 stream);
  }

  void SetUp() override {
    params =
      ::testing::TestWithParam<DistancesInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    make_data();

    Distance::distances_config_t<value_idx, value_t> dist_config;
    dist_config.b_nrows = 4;
    dist_config.b_ncols = 2;
    dist_config.b_nnz = 8;
    dist_config.b_indptr = csc_indptr;
    dist_config.b_indices = csc_indices;
    dist_config.b_data = csc_data;
    dist_config.a_nrows = 4;
    dist_config.a_ncols = 2;
    dist_config.a_nnz = 8;
    dist_config.a_indptr = indptr;
    dist_config.a_indices = indices;
    dist_config.a_data = data;
    dist_config.handle = cusparseHandle;
    dist_config.allocator = alloc;
    dist_config.stream = stream;

    allocate(out_dists, 16);

    Distance::ip_distances_t<value_idx, value_t> compute_dists(dist_config);
    compute_dists.compute(out_dists);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(csc_indptr));
    CUDA_CHECK(hipFree(csc_indices));
    CUDA_CHECK(hipFree(out_dists));
    CUDA_CHECK(hipFree(out_dists_ref));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_dists_ref, out_dists, 16, Compare<value_t>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t cusparseHandle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // transposed input
  value_idx *csc_indptr, *csc_indices;
  value_t *csc_data;

  // output data
  value_t *out_dists, *out_dists_ref;

  DistancesInputs<value_idx, value_t> params;
};

const std::vector<DistancesInputs<int, float>> inputs_i32_f = {{}};
typedef DistancesTest<int, float> DistancesTestF;
TEST_P(DistancesTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(DistancesTests, DistancesTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Selection
};  // end namespace Sparse
};  // end namespace MLCommon
