/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <hipsparse.h>
#include <gtest/gtest.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <sparse/knn.cuh>
#include "test_utils.h"

namespace MLCommon {
namespace Sparse {
namespace Selection {

template <typename value_idx, typename value_t>
struct KNNInputs {};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const KNNInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class SparseKNNTest
  : public ::testing::TestWithParam<KNNInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = {0, 2, 4, 6, 8};
    std::vector<value_idx> indices_h = {0, 4, 0, 3, 0, 2, 0, 8};
    std::vector<value_t> data_h = {0.0f, 1.0f, 5.0f, 6.0f,
                                   5.0f, 6.0f, 0.0f, 1.0f};

    allocate(indptr, 5);
    allocate(indices, 8);
    allocate(data, 8);

    updateDevice(indptr, indptr_h.data(), indptr_h.size(), stream);
    updateDevice(indices, indices_h.data(), indices_h.size(), stream);
    updateDevice(data, data_h.data(), data_h.size(), stream);

    std::vector<value_t> out_dists_ref_h = {1,  0,  0, 61, 25, 0,
                                            61, 25, 0, 1,  0,  0};
    std::vector<value_idx> out_indices_ref_h = {0, 3, 1, 1, 2, 3,
                                                2, 1, 3, 3, 2, 0};

    allocate(out_indices_ref, out_indices_ref_h.size());
    allocate(out_dists_ref, out_dists_ref_h.size());

    updateDevice(out_indices_ref, out_indices_ref_h.data(),
                 out_indices_ref_h.size(), stream);
    updateDevice(out_dists_ref, out_dists_ref_h.data(), out_dists_ref_h.size(),
                 stream);

    allocate(out_dists, 4 * k);
    allocate(out_indices, 4 * k);
  }

  /**
   * void brute_force_knn(
  const value_idx *idxIndptr, const value_idx *idxIndices,
  const value_t *idxData, value_idx idxNNZ, value_idx n_idx_rows,
  value_idx n_idx_cols, const value_idx *queryIndptr,
  const value_idx *queryIndices, const value_t *queryData, size_t queryNNZ,
  value_idx n_query_rows, value_idx n_query_cols, value_idx *output_indices,
  value_t *output_dists, int k, hipsparseHandle_t cusparseHandle,
  std::shared_ptr<deviceAllocator> allocator, hipStream_t stream,
  size_t batch_size = 2 << 20,  // approx 1M
  ML::MetricType metric = ML::MetricType::METRIC_L2, float metricArg = 0,
  bool expanded_form = false)
   */

  void SetUp() override {
    params =
      ::testing::TestWithParam<KNNInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(new defaultDeviceAllocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    k = 3;

    make_data();

    brute_force_knn<value_idx, value_t>(
      indptr, indices, data, 8, 4, 9, indptr, indices, data, 8, 4, 9,
      out_indices, out_dists, k, cusparseHandle, alloc, stream, 2 << 20,
      ML::MetricType::METRIC_INNER_PRODUCT);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_indices));
    CUDA_CHECK(hipFree(out_dists));
  }

  void compare() {
    ASSERT_TRUE(
      devArrMatch(out_dists_ref, out_dists, 4 * k, Compare<value_t>()));
    ASSERT_TRUE(
      devArrMatch(out_indices_ref, out_indices, 4 * k, Compare<value_idx>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t cusparseHandle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_idx *out_indices;
  value_t *out_dists;

  int k;

  value_idx *out_indices_ref;
  value_t *out_dists_ref;

  KNNInputs<value_idx, value_t> params;
};

const std::vector<KNNInputs<int, float>> inputs_i32_f = {{}};
typedef SparseKNNTest<int, float> KNNTestF;
TEST_P(KNNTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(SparseKNNTest, KNNTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Selection
};  // end namespace Sparse
};  // end namespace MLCommon
