/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/cudart_utils.h>
#include <hipsparse.h>
#include <common/device_buffer.hpp>

#include <gtest/gtest.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <test_utils.h>
#include <sparse/distances.cuh>

namespace MLCommon {
namespace Sparse {
namespace Selection {

template <typename value_idx, typename value_t>
struct L2DistancesInputs {};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const L2DistancesInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class L2DistancesTest
  : public ::testing::TestWithParam<L2DistancesInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = {0, 2, 4, 6, 8};
    std::vector<value_idx> indices_h = {0, 1, 0, 1, 0, 1, 0, 1};
    std::vector<value_t> data_h = {1.0f, 2.0f, 1.0f, 2.0f,
                                   1.0f, 2.0f, 1.0f, 2.0f};

    allocate(indptr, 5);
    allocate(indices, 8);
    allocate(data, 8);

    updateDevice(indptr, indptr_h.data(), 5, stream);
    updateDevice(indices, indices_h.data(), 8, stream);
    updateDevice(data, data_h.data(), 8, stream);

    std::vector<value_t> out_dists_ref_h = {
      0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
    };

    allocate(out_dists_ref, 16);

    updateDevice(out_dists_ref, out_dists_ref_h.data(), out_dists_ref_h.size(),
                 stream);
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      L2DistancesInputs<value_idx, value_t>>::GetParam();
    std::shared_ptr<deviceAllocator> alloc(
      new raft::mr::device::default_allocator);
    CUDA_CHECK(hipStreamCreate(&stream));

    CUSPARSE_CHECK(hipsparseCreate(&cusparseHandle));

    make_data();

    Distance::distances_config_t<value_idx, value_t> dist_config;
    dist_config.b_nrows = 4;
    dist_config.b_ncols = 2;
    dist_config.b_nnz = 8;
    dist_config.b_indptr = indptr;
    dist_config.b_indices = indices;
    dist_config.b_data = data;
    dist_config.a_nrows = 4;
    dist_config.a_ncols = 2;
    dist_config.a_nnz = 8;
    dist_config.a_indptr = indptr;
    dist_config.a_indices = indices;
    dist_config.a_data = data;
    dist_config.handle = cusparseHandle;
    dist_config.allocator = alloc;
    dist_config.stream = stream;

    allocate(out_dists, 16);

    ML::Logger::get().setLevel(CUML_LEVEL_DEBUG);


    pairwiseDistance(out_dists, dist_config, ML::Distance::DistanceType::EucExpandedL2);

    CUDA_CHECK(hipStreamSynchronize(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_dists));
    CUDA_CHECK(hipFree(out_dists_ref));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_dists_ref, out_dists, 16, Compare<value_t>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t cusparseHandle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_t *out_dists, *out_dists_ref;

  L2DistancesInputs<value_idx, value_t> params;
};

const std::vector<L2DistancesInputs<int, float>> inputs_i32_f = {{}};
typedef L2DistancesTest<int, float> L2DistancesTestF;
TEST_P(L2DistancesTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(L2DistancesTests, L2DistancesTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace Selection
};  // end namespace Sparse
};  // end namespace MLCommon
