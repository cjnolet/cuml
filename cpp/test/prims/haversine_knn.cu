/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/distance_type.h>
#include <iostream>
#include <raft/cuda_utils.cuh>
#include <selection/haversine_knn.cuh>
#include <vector>
#include "test_utils.h"

namespace MLCommon {
namespace Selection {

/**
 *
 * NOTE: Not exhaustively testing the kNN implementation since
 * we are using FAISS for this. Just testing API to verify the
 * knn.cu class is accepting inputs and providing outputs as
 * expected.
 */
template <typename value_idx, typename value_t>
class HaversineKNNTest : public ::testing::Test {
 protected:
  void basicTest() {
    auto alloc = std::make_shared<raft::mr::device::default_allocator>();

    // Allocate input
    raft::allocate(d_train_inputs, n * d);

    // Allocate reference arrays
    raft::allocate<value_idx>(d_ref_I, n * n);
    raft::allocate(d_ref_D, n * n);

    // Allocate predicted arrays
    raft::allocate<value_idx>(d_pred_I, n * n);
    raft::allocate(d_pred_D, n * n);

    // make testdata on host
    std::vector<value_t> h_train_inputs = {1.0, 50.0, 51.0};
    h_train_inputs.resize(n);
    raft::update_device(d_train_inputs, h_train_inputs.data(), n * d, 0);

    std::vector<value_t> h_res_D = {0.0,  49.0, 50.0, 0.0, 1.0,
                                    49.0, 0.0,  1.0,  50.0};
    h_res_D.resize(n * n);
    raft::update_device(d_ref_D, h_res_D.data(), n * n, 0);

    std::vector<value_idx> h_res_I = {0, 1, 2, 1, 2, 0, 2, 1, 0};
    h_res_I.resize(n * n);
    raft::update_device<value_idx>(d_ref_I, h_res_I.data(), n * n, 0);

    std::vector<value_t *> input_vec = {d_train_inputs};
    std::vector<value_idx> sizes_vec = {n};

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    raft::selection::haversine_knn(d_pred_I, d_pred_D, d_train_inputs,
                                   d_train_inputs, n, n, k, stream);

    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void SetUp() override { basicTest(); }

  void TearDown() override {
    CUDA_CHECK(hipFree(d_train_inputs));
    CUDA_CHECK(hipFree(d_pred_I));
    CUDA_CHECK(hipFree(d_pred_D));
    CUDA_CHECK(hipFree(d_ref_I));
    CUDA_CHECK(hipFree(d_ref_D));
  }

 protected:
  value_t *d_train_inputs;

  int n = 3;
  int d = 1;

  int k = 4;

  value_idx *d_pred_I;
  value_t *d_pred_D;

  value_idx *d_ref_I;
  value_t *d_ref_D;
};

typedef HaversineKNNTest<int, float> HaversineKNNTestF;

TEST_F(HaversineKNNTestF, Fit) {
  ASSERT_TRUE(raft::devArrMatch(d_ref_D, d_pred_D, n * n,
                                raft::CompareApprox<float>(1e-3)));
  ASSERT_TRUE(
    raft::devArrMatch(d_ref_I, d_pred_I, n * n, raft::Compare<int>()));
}

};  // end namespace Selection
};  // namespace MLCommon
