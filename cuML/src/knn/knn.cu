/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_utils.h"
#include "knn.h"
#include <hip/hip_runtime.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/Heap.h>
#include <faiss/gpu/GpuDistance.h>

#include <omp.h>
#include <vector>
#include <sstream>


namespace ML {


	/**
	 * Build a kNN object for training and querying a k-nearest neighbors model.
	 * @param D 	number of features in each vector
	 */
	kNN::kNN(const cumlHandle &handle, int D, bool verbose):
	        handle(handle.getImpl()),
	        D(D),
	        total_n(0),
	        indices(0),
	        verbose(verbose),
	        owner(false){}

	kNN::~kNN() {

	    try {
	        if(this->owner) {
	            if(this->verbose)
	                std::cout << "Freeing kNN memory" << std::endl;
	            for(kNNParams p : knn_params) { CUDA_CHECK(hipFree(p.ptr)); }
	        }

	    } catch(const std::exception &e) {
	        // cannot throw exception in destructor
	        std::cout << "An exception occurred releasing kNN memory: " << e.what() << std::endl;
	    }
	}

	void kNN::reset() {
        if(knn_params.size() > 0) {
            knn_params.clear();
            this->id_ranges.clear();
            this->indices = 0;
            this->total_n = 0;
        }
	}

	bool kNN::verify_size(size_t size, int device) {
        size_t free, total;
        hipMemGetInfo(&free, &total);

        if(size > free) {
            std::cout << "Not enough free memory on device "
                    << device
                    << " to run kneighbors. "
                    << "needed="
                    << size
                    << ", free=" << free << std::endl;
            return false;
        }

        return true;
	}

	/**
	 * Fit a kNN model by creating separate indices for multiple given
	 * instances of kNNParams.
	 * @param input  an array of pointers to data on (possibly different) devices
	 * @param N 	 number of items in input array.
	 */
	void kNN::fit(kNNParams *input, int N) {

        if(this->owner) {
            for(kNNParams p : knn_params) { CUDA_CHECK(hipFree(p.ptr)); }
        }

	    if(this->verbose)
	        std::cout << "N=" << N << std::endl;

	    reset();

        for(int i = 0; i < N; i++) {

            kNNParams params = input[i];

            hipPointerAttribute_t s_att;
            hipError_t s_err = hipPointerGetAttributes(&s_att, params.ptr);

            if(s_err != 0 || s_att.device == -1)
                throw std::runtime_error("Invalid device pointer encountered in knn fit()");


            this->indices++;
            this->knn_params.emplace_back(params);
            if(i < params.N) {
                id_ranges.push_back(total_n);
            }

            this->total_n += params.N;
        }
	}

	void cleanup(std::vector<faiss::gpu::StandardGpuResources *> dev_res, int indices) {
        for(int i = 0; i < indices; i++)
            delete dev_res[i];
	}


	/**
	 * Search the kNN for the k-nearest neighbors of a set of query vectors
	 * @param search_items set of vectors to query for neighbors
	 * @param n 		   number of items in search_items
	 * @param res_I 	   pointer to device memory for returning k nearest indices
	 * @param res_D		   pointer to device memory for returning k nearest distances
	 * @param k			   number of neighbors to query
	 */
	void kNN::search(const float *search_items, int n,
			long *res_I, float *res_D, int k) {

        hipPointerAttribute_t s_att;
        hipError_t s_err = hipPointerGetAttributes(&s_att, search_items);

        if(s_err != 0 || s_att.device == -1)
            throw std::runtime_error("Invalid device pointer encountered in knn search()");

        s_err = hipPointerGetAttributes(&s_att, res_I);

        if(s_err != 0 || s_att.device == -1)
            throw std::runtime_error("Invalid index results pointer encountered in knn search()");

        s_err = hipPointerGetAttributes(&s_att, res_D);

        if(s_err != 0 || s_att.device == -1)
            throw std::runtime_error("Invalid distance results pointer encountered in knn search()");

        hipStream_t *streams = new hipStream_t[indices];
        int *devices = new int[indices];
        std::vector<faiss::gpu::StandardGpuResources*> dev_res;


        /**
         * Verify all memory is on device,
         */

		for(int i = 0; i < indices; i++) {

            kNNParams params = knn_params[i];

            hipPointerAttribute_t att;
            hipError_t err = hipPointerGetAttributes(&att, params.ptr);

            if(err == 0 && att.device > -1) {

                devices[i] = att.device;

                CUDA_CHECK(hipSetDevice(att.device));

                if(!verify_size(size_t(params.N)*size_t(this->D)*4l, att.device)) {
                    delete streams;
                    delete devices;
                    cleanup(dev_res, indices);
                    return;
                }

                hipStream_t stream;
                hipStreamCreate(&stream);

                streams[i] = stream;

                dev_res.emplace_back(new faiss::gpu::StandardGpuResources());
                dev_res[i]->noTempMemory();
                dev_res[i]->setCudaMallocWarning(false);
                dev_res[i]->setDefaultStream(att.device, stream);

            } else {

                delete streams;
                delete devices;
                cleanup(dev_res, indices);

                std::stringstream ss;
                ss << "Input memory for " << &params << " failed. isDevice?=" << att.devicePointer << ", N=" << params.N;
                throw std::runtime_error(ss.str());

            }
		}

		hipStream_t stream = handle.getStream();

		MLCommon::host_buffer<float> all_D(handle.getHostAllocator(), stream, indices*k*size_t(n));
        MLCommon::host_buffer<long> all_I(handle.getHostAllocator(), stream, indices*k*size_t(n));

		/**
		 * Perform search in multiple threads / streams
		 */
        #pragma omp parallel
		{
            #pragma omp for
            for(int i = 0; i < indices; i++) {

                kNNParams params = knn_params[i];

                faiss::gpu::StandardGpuResources *gpu_res = dev_res[i];
                CUDA_CHECK(hipSetDevice(devices[i]));

                try {

                    bruteForceKnn(gpu_res,
                                faiss::METRIC_L2,
                                params.ptr,
                                params.N,
                                search_items,
                                n,
                                this->D,
                                k,
                                all_D.begin()+(long(i)*k*long(n)),
                                all_I.begin()+(long(i)*k*long(n)));

                    CUDA_CHECK(hipPeekAtLastError());
                    CUDA_CHECK(hipStreamSynchronize(streams[i]));

                } catch(const std::exception &e) {
                   std::cout << "Exception occurred in multi-threaded kNN search: " << e.what() << std::endl;
                }
            }
		}

		for(int i = 0; i < indices; i++)
	        hipStreamDestroy(streams[i]);

        MLCommon::host_buffer<float> result_D(handle.getHostAllocator(), stream, k*size_t(n));
        MLCommon::host_buffer<long> result_I(handle.getHostAllocator(), stream, k*size_t(n));

		merge_tables<faiss::CMin<float, int>>(long(n), k, indices,
				result_D.begin(), result_I.begin(), all_D.begin(), all_I.begin(), id_ranges.data());

		MLCommon::updateDevice(res_D, result_D.begin(), result_D.size(), 0);
		MLCommon::updateDevice(res_I, result_I.begin(), result_D.size(), 0);

        cleanup(dev_res, indices);
        delete streams;

		all_D.release(stream);
		all_I.release(stream);

		result_D.release(stream);
		result_I.release(stream);
	}

    /**
     * Chunk a host array up into one or many GPUs (determined by the provided
     * list of gpu ids) and fit a knn model.
     *
     * @param ptr       an array in host memory to chunk over devices
     * @param n         number of elements in ptr
     * @param devices   array of device ids for chunking the ptr
     * @param n_chunks  number of elements in gpus
     * @param out       host pointer (size n) to store output
     */
    void kNN::fit_from_host(float *ptr, int n, int* devices, int n_chunks) {

        if(this->owner) {
            for(kNNParams p : knn_params) { CUDA_CHECK(hipFree(p.ptr)); }
        }

        reset();

        size_t chunk_size = MLCommon::ceildiv<size_t>((size_t)n, (size_t)n_chunks);
        kNNParams params[n_chunks];

        this->owner = true;

        /**
         * Initial verification of memory
         */
        for(int i = 0; i < n_chunks; i++) {

            int device = devices[i];
            size_t length = chunk_size;
            if(length * i >= n)
                length = (chunk_size*i)-size_t(n);
            CUDA_CHECK(hipSetDevice(device));
            if(!verify_size(size_t(length)*size_t(D), device))
                return;
        }

        #pragma omp parallel for
        for(int i = 0; i < n_chunks; i++) {

            int device = devices[i];
            CUDA_CHECK(hipSetDevice(device));

            size_t length = chunk_size;
            if(length * i >= n)
                length = (size_t(chunk_size)*i)-size_t(n);

            float *ptr_d;
            MLCommon::allocate(ptr_d, size_t(length)*size_t(D));
            MLCommon::updateDevice(ptr_d, ptr+(size_t(chunk_size)*i), size_t(length)*size_t(D), 0);

            kNNParams p;
            p.N = length;
            p.ptr = ptr_d;

            params[i] = p;
        }

        fit(params, n_chunks);
   }

	/** Merge results from several shards into a single result set.
	 * @param all_distances  size nshard * n * k
	 * @param all_labels     idem
	 * @param translartions  label translations to apply, size nshard
	 */
	template <class C>
	void kNN::merge_tables (long n, long k, long nshard,
					   float *distances, long *labels,
					   float *all_distances,
					   long *all_labels,
					   long *translations) {
		if(k == 0) {
			return;
		}

		size_t stride = n * k;
		#pragma omp parallel
		{
			std::vector<int> buf (2 * nshard);
			int * pointer = buf.data();
			int * shard_ids = pointer + nshard;
			std::vector<float> buf2 (nshard);
			float * heap_vals = buf2.data();
			#pragma omp for
			for (long i = 0; i < n; i++) {
				// the heap maps values to the shard where they are
				// produced.
				const float *D_in = all_distances + i * k;
				const long *I_in = all_labels + i * k;
				int heap_size = 0;

				for (long s = 0; s < nshard; s++) {
					pointer[s] = 0;
					if (I_in[stride * s] >= 0)
						heap_push<C> (++heap_size, heap_vals, shard_ids,
									 D_in[stride * s], s);
				}

				float *D = distances + i * k;
				long *I = labels + i * k;

				for (int j = 0; j < k; j++) {
					if (heap_size == 0) {
						I[j] = -1;
						D[j] = C::neutral();
					} else {
						// pop best element
						int s = shard_ids[0];
						int & p = pointer[s];
						D[j] = heap_vals[0];
						I[j] = I_in[stride * s + p] + translations[s];

						heap_pop<C> (heap_size--, heap_vals, shard_ids);
						p++;
						if (p < k && I_in[stride * s + p] >= 0)
							heap_push<C> (++heap_size, heap_vals, shard_ids,
										 D_in[stride * s + p], s);
					}
				}
			}
		}
	};

};


// end namespace ML
