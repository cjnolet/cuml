/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_utils.h"
#include "knn.h"
#include <hip/hip_runtime.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/Heap.h>
#include <faiss/gpu/GpuDistance.h>

#include <memory>
#include <omp.h>
#include <vector>
#include <sstream>


namespace ML {


	/**
	 * Build a kNN object for training and querying a k-nearest neighbors model.
	 * @param D 	number of features in each vector
	 */
	kNN::kNN(const cumlHandle &handle, int D, bool verbose):
	        handle(handle.getImpl()),
	        D(D),
	        total_n(0),
	        indices(0),
	        verbose(verbose),
	        owner(false){}

	kNN::~kNN() {

	    try {
	        if(this->owner) {
	            if(this->verbose)
	                std::cout << "Freeing kNN memory" << std::endl;
	            for(kNNParams p : knn_params) { CUDA_CHECK(hipFree(p.ptr)); }
	        }

	    } catch(const std::exception &e) {
	        // cannot throw exception in destructor
	        std::cout << "An exception occurred releasing kNN memory: " << e.what() << std::endl;
	    }
	}

	void kNN::reset() {
        if(knn_params.size() > 0) {
            knn_params.clear();
            this->id_ranges.clear();
            this->indices = 0;
            this->total_n = 0;
        }
	}

	bool verify_size(size_t size, int device) {
        size_t free, total;
        hipMemGetInfo(&free, &total);

        if(size > free) {
            std::cout << "Not enough free memory on device "
                    << device
                    << " to run kneighbors. "
                    << "needed="
                    << size
                    << ", free=" << free << std::endl;
            return false;
        }

        return true;
	}


    template<typename T>
     void ASSERT_DEVICE_MEM(T *mem, std::string memName) {

        hipPointerAttribute_t s_att;
        hipError_t s_err = hipPointerGetAttributes(&s_att, mem);

        if(s_err != 0 || s_att.device == -1) {
            std::stringstream ss;
            ss << "Invalid device pointer encountered in knn (" <<
                    memName << ")" << ", error=" << s_err << std::endl;
            throw std::runtime_error(ss.str());
        }
    }

	/**
	 * Fit a kNN model by creating separate indices for multiple given
	 * instances of kNNParams.
	 * @param input  an array of pointers to data on (possibly different) devices
	 * @param N 	 number of items in input array.
	 */
	void kNN::fit(kNNParams *input, size_t N) {

        if(this->owner) {
            for(kNNParams p : knn_params) { CUDA_CHECK(hipFree(p.ptr)); }
        }

	    if(this->verbose)
	        std::cout << "N=" << N << std::endl;

	    reset();

        for(size_t i = 0; i < N; i++) {

            kNNParams params = input[i];

            hipPointerAttribute_t s_att;
            hipError_t s_err = hipPointerGetAttributes(&s_att, params.ptr);

            ASSERT_DEVICE_MEM(params.ptr, "ptr");

            this->indices++;
            this->knn_params.emplace_back(params);
            if(i < params.N) {
                id_ranges.push_back(total_n);
            }

            this->total_n += params.N;
        }
	}



	/**
	 * Search the kNN for the k-nearest neighbors of a set of query vectors
	 * @param search_items set of vectors to query for neighbors
	 * @param n 		   number of items in search_items
	 * @param res_I 	   pointer to device memory for returning k nearest indices
	 * @param res_D		   pointer to device memory for returning k nearest distances
	 * @param k			   number of neighbors to query
	 */
	void kNN::search(const float *search_items, size_t n,
			long *res_I, float *res_D, size_t k) {


	    ASSERT_DEVICE_MEM(search_items, "search_items");
	    ASSERT_DEVICE_MEM(res_I, "res_I");
	    ASSERT_DEVICE_MEM(res_D, "res_D");

		hipStream_t main_stream = handle.getStream();

		std::cout << "Stream=" << main_stream << std::endl;

		MLCommon::host_buffer<float> all_D(handle.getHostAllocator(), main_stream, indices*k*size_t(n));
        MLCommon::host_buffer<long> all_I(handle.getHostAllocator(), main_stream, indices*k*size_t(n));

        std::cout << "Searching in parallel" << std::endl;

		/**
		 * Perform search in multiple threads / streams
		 */
        #pragma omp parallel
		{
            #pragma omp for
            for(size_t i = 0; i < indices; i++) {

                kNNParams params = knn_params[i];

                hipPointerAttribute_t att;
                hipError_t s_err = hipPointerGetAttributes(&att, params.ptr);

                CUDA_CHECK(hipSetDevice(att.device));

                hipStream_t stream;
                hipStreamCreate(&stream);

                faiss::gpu::StandardGpuResources gpu_res;
                gpu_res.noTempMemory();
                gpu_res.setCudaMallocWarning(false);
                gpu_res.setDefaultStream(att.device, stream);

                try {
                    bruteForceKnn(&gpu_res,
                                faiss::METRIC_L2,
                                params.ptr,
                                params.N,
                                search_items,
                                n,
                                this->D,
                                k,
                                all_D.data()+(i*k*n),
                                all_I.data()+(i*k*n));

                    std::cout << "DONE!" << std::endl;

                    CUDA_CHECK(hipPeekAtLastError());
                    CUDA_CHECK(hipStreamSynchronize(stream));
                    hipStreamDestroy(stream);

                } catch(const std::exception &e) {
                   std::cout << "Exception occurred in multi-threaded kNN search: " << e.what() << std::endl;
                }
            }
		}

		std::cout << "END!" << std::endl;

        MLCommon::host_buffer<float> result_D(handle.getHostAllocator(), main_stream, k*size_t(n));
        MLCommon::host_buffer<long> result_I(handle.getHostAllocator(), main_stream, k*size_t(n));

		merge_tables<faiss::CMin<float, int>>(long(n), k, indices,
				result_D.begin(), result_I.begin(), all_D.begin(), all_I.begin(),
				id_ranges.data());

		MLCommon::updateDevice(res_D, result_D.begin(), result_D.size(), main_stream);
		MLCommon::updateDevice(res_I, result_I.begin(), result_D.size(), main_stream);

		all_D.release(main_stream);
		all_I.release(main_stream);

		result_D.release(main_stream);
		result_I.release(main_stream);
	}

    /**
     * Chunk a host array up into one or many GPUs (determined by the provided
     * list of gpu ids) and fit a knn model.
     *
     * @param ptr       an array in host memory to chunk over devices
     * @param n         number of elements in ptr
     * @param devices   array of device ids for chunking the ptr
     * @param n_chunks  number of elements in gpus
     * @param out       host pointer (size n) to store output
     */
    void kNN::fit_from_host(float *ptr, size_t n, int* devices, size_t n_chunks) {

        if(this->owner) {
            for(kNNParams p : knn_params) { CUDA_CHECK(hipFree(p.ptr)); }
        }

        reset();

        size_t chunk_size = MLCommon::ceildiv<size_t>(
                (size_t)n, (size_t)n_chunks);
        kNNParams params[n_chunks];

        this->owner = true;

        /**
         * Initial verification of memory
         */
        for(int i = 0; i < n_chunks; i++) {

            int device = devices[i];
            size_t length = chunk_size;
            if(length * i >= n)
                length = (chunk_size*i)-size_t(n);
            CUDA_CHECK(hipSetDevice(device));
            if(!verify_size(size_t(length)*size_t(D), device))
                return;
        }

        for(int i = 0; i < n_chunks; i++) {

            int device = devices[i];
            CUDA_CHECK(hipSetDevice(device));

            size_t length = chunk_size;
            if(length * i >= n)
                length = (size_t(chunk_size)*i)-size_t(n);

            float *ptr_d;
            MLCommon::allocate(ptr_d, size_t(length)*size_t(D));
            MLCommon::updateDevice(ptr_d, ptr+(size_t(chunk_size)*i),
                    size_t(length)*size_t(D), this->handle.getStream());

            kNNParams p;
            p.N = length;
            p.ptr = ptr_d;

            params[i] = p;
        }

        std::cout << "Fitting!" << std::endl;
        fit(params, n_chunks);
   }

	/** Merge results from several shards into a single result set.
	 * @param all_distances  size nshard * n * k
	 * @param all_labels     idem
	 * @param translartions  label translations to apply, size nshard
	 */
	template <class C>
	void kNN::merge_tables (long n, long k, long nshard,
					   float *distances, long *labels,
					   float *all_distances,
					   long *all_labels,
					   long *translations) {
		if(k == 0) {
			return;
		}

		size_t stride = n * k;
		#pragma omp parallel
		{
			std::vector<int> buf (2 * nshard);
			int * pointer = buf.data();
			int * shard_ids = pointer + nshard;
			std::vector<float> buf2 (nshard);
			float * heap_vals = buf2.data();
			#pragma omp for
			for (long i = 0; i < n; i++) {
				// the heap maps values to the shard where they are
				// produced.
				const float *D_in = all_distances + i * k;
				const long *I_in = all_labels + i * k;
				int heap_size = 0;

				for (long s = 0; s < nshard; s++) {
					pointer[s] = 0;
					if (I_in[stride * s] >= 0)
						heap_push<C> (++heap_size, heap_vals, shard_ids,
									 D_in[stride * s], s);
				}

				float *D = distances + i * k;
				long *I = labels + i * k;

				for (int j = 0; j < k; j++) {
					if (heap_size == 0) {
						I[j] = -1;
						D[j] = C::neutral();
					} else {
						// pop best element
						int s = shard_ids[0];
						int & p = pointer[s];
						D[j] = heap_vals[0];
						I[j] = I_in[stride * s + p] + translations[s];

						heap_pop<C> (heap_size--, heap_vals, shard_ids);
						p++;
						if (p < k && I_in[stride * s + p] >= 0)
							heap_push<C> (++heap_size, heap_vals, shard_ids,
										 D_in[stride * s + p], s);
					}
				}
			}
		}
	};

};


// end namespace ML
